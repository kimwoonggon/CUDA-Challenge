#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hipblaslt.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <cmath>
#include <algorithm>

#define N 1024
#define TILE 16

__global__ void init(float *x, unsigned int seed)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N && col < N)
    {
        hiprandState state;
        hiprand_init(seed, row * N + col, 0, &state);
        x[row * N + col] = hiprand_uniform(&state) * 2.0f - 1.0f;
    }
}

__global__ void matMulFusedTiledQuant(float *a, float *b, half *c, float scale_a, float scale_b)
{
    __shared__ int8_t tile_a[TILE][TILE];
    __shared__ int8_t tile_b[TILE][TILE];

    int row = blockIdx.y * TILE + threadIdx.y;
    int col = blockIdx.x * TILE + threadIdx.x;
    int ty = threadIdx.y;
    int tx = threadIdx.x;

    int32_t acc = 0;

    for (int m = 0; m < N / TILE; m++)
    {
        if (row < N && (m * TILE + tx) < N)
        {
            float val_a = a[row * N + m * TILE + tx];
            tile_a[ty][tx] = max(-128, min(127, __float2int_rn(val_a / scale_a)));
        }
        else
            tile_a[ty][tx] = 0;

        if (col < N && (m * TILE + ty) < N)
        {
            float val_b = b[(m * TILE + ty) * N + col];
            tile_b[ty][tx] = max(-128, min(127, __float2int_rn(val_b / scale_b)));
        }
        else
            tile_b[ty][tx] = 0;

        __syncthreads();

        for (int k = 0; k < TILE; k++)
        {
            acc += (int32_t)tile_a[ty][k] * (int32_t)tile_b[k][tx];
        }

        __syncthreads();
    }

    if (row < N && col < N)
    {
        float val = acc * scale_a * scale_b;
        c[row * N + col] = __float2half(val);
    }
}

void runCuBLASLtINT8(const int8_t *a_i8, const int8_t *b_i8, half *c_fp16)
{
    hipblasLtHandle_t ltHandle;
    hipblasLtMatmulDesc_t opDesc;
    hipblasLtMatrixLayout_t aLayout, bLayout, cLayout;
    hipblasLtMatmulPreference_t preference;
    size_t workspaceSize = 4 * 1024 * 1024;
    void* workspace;
    hipMalloc(&workspace, workspaceSize);

    hipblasLtCreate(&ltHandle);

    hipblasLtMatmulDescCreate(&opDesc, HIPBLAS_COMPUTE_32I, HIP_R_32I);
    hipblasLtMatrixLayoutCreate(&aLayout, HIP_R_8I, N, N, N);
    hipblasLtMatrixLayoutCreate(&bLayout, HIP_R_8I, N, N, N);
    hipblasLtMatrixLayoutCreate(&cLayout, HIP_R_16F, N, N, N);

    hipblasLtMatmulPreferenceCreate(&preference);
    hipblasLtMatmulPreferenceSetAttribute(preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize));

    int32_t alpha = 1;
    int32_t beta = 0;

    hipblasLtMatmul(ltHandle, opDesc, &alpha, a_i8, aLayout, b_i8, bLayout, &beta, c_fp16, cLayout, c_fp16, cLayout, NULL, workspace, workspaceSize, 0);

    hipblasLtDestroy(ltHandle);
    hipFree(workspace);
}

int main()
{
    float *a_fp32, *b_fp32;
    int8_t *a_int8, *b_int8;
    half *c_fused, *c_cublas;

    size_t size_fp32 = N * N * sizeof(float);
    size_t size_int8 = N * N * sizeof(int8_t);
    size_t size_fp16 = N * N * sizeof(half);

    hipMallocManaged(&a_fp32, size_fp32);
    hipMallocManaged(&b_fp32, size_fp32);
    hipMallocManaged(&a_int8, size_int8);
    hipMallocManaged(&b_int8, size_int8);
    hipMallocManaged(&c_fused, size_fp16);
    hipMallocManaged(&c_cublas, size_fp16);

    dim3 threads(TILE, TILE);
    dim3 blocks((N + TILE - 1) / TILE, (N + TILE - 1) / TILE);

    init<<<blocks, threads>>>(a_fp32, time(NULL));
    init<<<blocks, threads>>>(b_fp32, time(NULL));
    hipDeviceSynchronize();

    float scale_a = 0.02f;
    float scale_b = 0.02f;

    for (int i = 0; i < N * N; i++)
    {
        a_int8[i] = static_cast<int8_t>(std::max(-128, std::min(127, static_cast<int>(std::round(a_fp32[i] / scale_a)))));
        b_int8[i] = static_cast<int8_t>(std::max(-128, std::min(127, static_cast<int>(std::round(b_fp32[i] / scale_b)))));
    }

    float fused_time = 0.0f;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    matMulFusedTiledQuant<<<blocks, threads>>>(a_fp32, b_fp32, c_fused, scale_a, scale_b);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&fused_time, start, stop);

    float cublas_time = 0.0f;
    hipEventRecord(start);
    runCuBLASLtINT8(a_int8, b_int8, c_cublas);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cublas_time, start, stop);

    printf("Fused Tiled (int8→fp16): %.2f ms\n", fused_time);
    printf("cuBLASLt INT8 GEMM: %.2f ms\n", cublas_time);

    hipFree(a_fp32);
    hipFree(b_fp32);
    hipFree(a_int8);
    hipFree(b_int8);
    hipFree(c_fused);
    hipFree(c_cublas);

    return 0;
}
